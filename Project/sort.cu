// using constant memeory


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
using namespace std;

#define NUM_RANGE 101

//INSERT CODE HERE---------------------------------
__global__ void histogram(unsigned int *hist, int *pSource, int input_size)
{
	__shared__ int histShared[NUM_RANGE];
	if(threadIdx.x < NUM_RANGE)
		histShared[threadIdx.x] = 0;

	__syncthreads();
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < input_size)
	{
		int pixelVal = pSource[i];
		atomicAdd(&(histShared[pixelVal]), 1);
	}

	__syncthreads();
	if (threadIdx.x < NUM_RANGE)
		atomicAdd(&(hist[threadIdx.x]), histShared[threadIdx.x]);
}

__global__ void prefix(unsigned int *hist, unsigned int *pre)
{
	int x = threadIdx.x;
	int num = blockDim.x;
	__shared__ unsigned int histShared[128];
	if(x < NUM_RANGE)	histShared[x] = hist[x];
	else				histShared[x] = 0;

	int stride = 1;
	while(stride < num)
	{
		int index = (x + 1) * stride * 2 - 1;
		if(index < num)
			histShared[index] += histShared[index - stride];
		stride *= 2;

		__syncthreads();
	}

	stride = num / 2;
	while(stride > 0)
	{
		int index = (x + 1) * stride * 2 - 1;
		if(index < num && (index+stride) < num)
			histShared[index+stride] += histShared[index];
		stride /= 2;
		__syncthreads();
	}

	if(x < NUM_RANGE)
		pre[x] = histShared[x];	
}

__global__ void fill_matrix(int *result, unsigned int *pre)
{
	int i = threadIdx.x;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ unsigned int preShared[NUM_RANGE + 1];
	if(i == 0)					preShared[i] = 0;
	else if(i < NUM_RANGE)		preShared[i] = pre[i-1];
	__syncthreads();

	int left = 0, right = NUM_RANGE, h = (left + right) / 2;
	while(left + 1 < right)
	{
		if(preShared[h] > x)			right = h;
		else							left = h;
		h = (left + right) / 2;
	}
	result[x] = h;
}

void verify(int* src, int*result, int input_size){
	sort(src, src+input_size);
	long long match_cnt=0;
	for(int i=0; i<input_size;i++)
	{
		if(src[i]==result[i])
			match_cnt++;
	}

	if(match_cnt==input_size)
		printf("TEST PASSED\n\n");
	else
		printf("TEST FAILED\n\n");

}

void genData(int* ptr, unsigned int size) {
	while (size--) {
		*ptr++ = (int)(rand() % 101);
	}
}

int main(int argc, char* argv[]) {
	int* pSource = NULL;
	int* pResult = NULL;
	int input_size=0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	if (argc == 2)
		input_size=atoi(argv[1]);
	else
	{
    		printf("\n    Invalid input parameters!"
	   		"\n    Usage: ./sort <input_size>"
           		"\n");
        	exit(0);
	}

	//allocate host memory
	pSource=(int*)malloc(input_size*sizeof(int));
	pResult=(int*)malloc(input_size*sizeof(int));
	// generate source data
	genData(pSource, input_size);
	
	// start timer
	hipEventRecord(start, 0);

	/////////////////////////////// histogram /////////////////////////////////////////////////////////////

	// allocate host memory
	unsigned int *hist;
	unsigned int *pre;
	hist = (unsigned int *)malloc(NUM_RANGE*sizeof(unsigned int));
	pre = (unsigned int *)malloc(NUM_RANGE*sizeof(unsigned int));
	for(int i=0;i<NUM_RANGE;i++)	hist[i] = 0;
	for(int i=0;i<NUM_RANGE;i++)	pre[i] = 0;

	// allocate device memory
	int *pSourcedev = NULL;
	int *pResultdev = NULL;
	unsigned int *histdev = NULL;
	unsigned int *predev = NULL;
	hipMalloc((void **)&pSourcedev, input_size * sizeof(int));
	hipMalloc((void **)&pResultdev, input_size * sizeof(int));
	hipMalloc((void **)&histdev, NUM_RANGE * sizeof(unsigned int));
	hipMalloc((void **)&predev, NUM_RANGE * sizeof(unsigned int));

	// copy from host to device
	hipMemcpy(pSourcedev, pSource, input_size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(histdev, hist, NUM_RANGE * sizeof(unsigned int), hipMemcpyHostToDevice);
	// cudaMemcpy(predev, pre, NUM_RANGE * sizeof(unsigned int), cudaMemcpyHostToDevice); // if prefix device kernel active

	// launch the kernel -> histogram
	int BLOCK_SIZE_HIST = 512;
	int GRID_SIZE_HIST = ceil(input_size / (float)BLOCK_SIZE_HIST);
	dim3 dimgrid_hist(GRID_SIZE_HIST, 1, 1);
	dim3 dimblock_hist(BLOCK_SIZE_HIST, 1, 1);
	histogram <<< dimgrid_hist, dimblock_hist >>> (histdev, pSourcedev, input_size);

	// copy from device to host
	hipMemcpy(hist, histdev, NUM_RANGE * sizeof(unsigned int), hipMemcpyDeviceToHost);

	//////////////////////////////////// prefix //////////////////////////////////////////////////////////////////////

	/****************** prefix device kernel **********************/
	// launch ther kernel -> prefix
	/*int BLOCK_SIZE_PRE = 128;
	int GRID_SIZE_PRE = 1;
	dim3 dimgrid_pre(GRID_SIZE_PRE, 1, 1);
	dim3 dimblock_pre(BLOCK_SIZE_PRE, 1, 1);
	prefix <<< dimgrid_pre, dimblock_pre >>> (histdev, predev);

	// copy from device to host
	cudaMemcpy(pre, predev, NUM_RANGE * sizeof(unsigned int), cudaMemcpyDeviceToHost);*/
	/***************************************************************/
	
	pre[0] = hist[0];
	for(int i=0;i<101;i++)
		pre[i] = pre[i-1] + hist[i];

	////////////////////////////////////// fill matrix ///////////////////////////////////////////////////////////////////

	// copy from host to device
	hipMemcpy(predev, pre, NUM_RANGE * sizeof(unsigned int), hipMemcpyHostToDevice); // if prefix device kernel not active

	// launch kernel
	int BLOCK_SIZE_FILL = 512;
	int GRID_SIZE_FILL = ceil(input_size / (float)BLOCK_SIZE_FILL);
	dim3 dimgrid_fill(GRID_SIZE_FILL, 1, 1);
	dim3 dimblock_fill(BLOCK_SIZE_FILL, 1, 1);
	fill_matrix <<< dimgrid_fill, dimblock_fill >>> (pResultdev, predev);

	hipMemcpy(pResult, pResultdev, input_size * sizeof(int), hipMemcpyDeviceToHost);

	////////////////////////////////////////////////////////////////////////////////////////////////////////////////


	// end timer
	float time;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("elapsed time = %f msec\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// test code ////////////////
	/*printf("hist: ");
	for(int i = 0;i<101;i++)
		printf("%u ", hist[i]);
	printf("\n");

	printf("pre: ");
	for(int i = 0;i<101;i++)
		printf("%d: %u ", i, pre[i]);
	printf("\n");

	printf("Result: ");
	for(int i = 0;i<=100;i++)
	{
		int j = 0;
		while(1)
		{
			if(i < pResult[j])
			{
				printf("%d: %d ", i, j);
				break;
			}
			j++;
		}
	}
	printf("\n");

	printf("pre: ");
	for(int i = 0;i<input_size;i++)
		printf("%d ", pResult[i]);
	printf("\n");*/
	/////////////////////////////

	printf("Verifying results..."); fflush(stdout);
	verify(pSource, pResult, input_size);
	fflush(stdout);
}
