#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>

#ifdef DEBUG
#define CUDA_CHECK(x) do {\
	(x);\
	hipError_t e = hipGetLastError();\
	if(hipSuccess !=e){\
		printf("cuda failure \"%s\" at %s:%d\n",\
			hipGetErrorString(e),\
			__FILE__,__LINE__);\
		exit(1);\
	}\
}while(0)
#else
#define CUDA_CHECK(x)	(x)
#endif

// kernel program for the device (GPU): compiled by NVCC
__global__ void addKernel(int* c, const int* a, const int* b, const int width) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int i = y * width + x; // [y][x] = y * WIDTH + x;
	c[i] = a[i] + b[i];
}

// main program for the CPU: compiled by MS-VC++
int main(int argc, char* argv[]) {

	// host-side data
	const int HEIGHT = atoi(argv[1]);
	const float TILE_HEIGHT = 16;
	const int WIDTH = atoi(argv[2]);
	const float TILE_WIDTH = 16;
	int a[HEIGHT][WIDTH];
	int b[HEIGHT][WIDTH];
	int c[HEIGHT][WIDTH] = {0};

	// make a, b matrices
	for (int y = 0; y < HEIGHT; ++y)
	{
		for (int x = 0; x < WIDTH; ++x)
		{
			a[y][x] = y * 10 + x;
			b[y][x] = (y * 10 + x) * 10000;
		}
	}

	// device-side data
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;

	// allocate device memory
	CUDA_CHECK( hipMalloc((void**)&dev_a, HEIGHT * WIDTH * sizeof(int)) );
	CUDA_CHECK( hipMalloc((void**)&dev_b, HEIGHT * WIDTH * sizeof(int)) );
	CUDA_CHECK( hipMalloc((void**)&dev_c, HEIGHT * WIDTH * sizeof(int)) );

	// copy from host to device
	CUDA_CHECK( hipMemcpy(dev_a, a, HEIGHT * WIDTH * sizeof(int), hipMemcpyHostToDevice) );
	CUDA_CHECK( hipMemcpy(dev_b, b, HEIGHT * WIDTH * sizeof(int), hipMemcpyHostToDevice) );

	// launch a kernel on the GPU with one thread for each element.
	dim3 dimGrid(ceil(WIDTH/TILE_WIDTH), ceil(HEIGHT/TILE_HEIGHT), 1);
	dim3 dimBlock(int(TILE_WIDTH), int(TILE_HEIGHT), 1); // x, y, z
	addKernel <<< dimGrid, dimBlock>>>(dev_c, dev_a, dev_b, WIDTH); // dev_c = dev_a + dev_b;
	CUDA_CHECK( hipPeekAtLastError() );

	// copy from device to host
	CUDA_CHECK( hipMemcpy(c, dev_c, HEIGHT * WIDTH * sizeof(int), hipMemcpyDeviceToHost) );

	// free device memory
	CUDA_CHECK( hipFree(dev_c) );
	CUDA_CHECK( hipFree(dev_a) );
	CUDA_CHECK( hipFree(dev_b) );

	// print the result
	for (int y = 0; y < HEIGHT; ++y) {
		for (int x = 0; x < WIDTH; ++x)
			printf("%8d", c[y][x]);
		printf("\n");
	}
	// done
	return 0;
}



