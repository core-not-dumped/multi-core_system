#include "hip/hip_runtime.h"
__constant__ float M[FILTER_SIZE][FILTER_SIZE];

__global__ void convolution(Matrix N, Matrix P)
{
    //INSERT KERNEL CODE HERE
	const int TILE_SIZE = 16;
	const int BLOCK_SIZE = TILE_SIZE + FILTER_SIZE - 1;
	__shared__ float N_ds[BLOCK_SIZE][BLOCK_SIZE];
	int ty = threadIdx.y;
	int tx = threadIdx.x;
	int row_o = blockIdx.y * TILE_SIZE + ty;
	int col_o = blockIdx.x * TILE_SIZE + tx;
	int row_i = row_o - FILTER_SIZE/2;
	int col_i = col_o - FILTER_SIZE/2;
	float output = 0.0f;

	if((row_i >= 0) && (row_i < N.height) && (col_i >= 0) && (col_i < N.width))
		N_ds[ty][tx] = N.elements[row_i * N.width + col_i];
	else
		N_ds[ty][tx] = 0.0f;

	__syncthreads();

	if(ty < TILE_SIZE && tx < TILE_SIZE)
	{
		for(size_t i = 0; i < FILTER_SIZE; i++)
		{
			for(size_t j = 0; j < FILTER_SIZE; j++)
				output += M[i][j] * N_ds[i + ty][j + tx];
		}
		if(row_o < P.height && col_o < P.width)
			P.elements[row_o * P.width + col_o] = output;
	}	
}
