#include "hip/hip_runtime.h"
#include <stdio.h>
#include "support.h"
#include "kernel.cu"

int main(int argc, char* argv[])
{
    Timer timer;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    Matrix M_h, N_h, P_h; // M: filter, N: input image, P: output image
    Matrix N_d, P_d;
    unsigned imageHeight, imageWidth;
    hipError_t cuda_ret;
    dim3 dim_grid, dim_block;

    /* Read image dimensions */
    if (argc == 3) {
        imageHeight = atoi(argv[1]);
        imageWidth = atoi(argv[2]);
    } else {
        printf("\n    Invalid input parameters!"
	   "\n    Usage: ./convolution <m> <n>  # Image is m x n"
           "\n");
        exit(0);
    }

    /* Allocate host memory */
    M_h = allocateMatrix(FILTER_SIZE, FILTER_SIZE);
    N_h = allocateMatrix(imageHeight, imageWidth);
    P_h = allocateMatrix(imageHeight, imageWidth);

    /* Initialize filter and images */
    initMatrix(M_h);
    initMatrix(N_h);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Image: %u x %u\n", imageHeight, imageWidth);
    printf("    Mask: %u x %u\n", FILTER_SIZE, FILTER_SIZE);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    N_d = allocateDeviceMatrix(imageHeight, imageWidth);
    P_d = allocateDeviceMatrix(imageHeight, imageWidth);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    /* Copy image to device global memory */
    //INSERT CODE HERE
	copyToDeviceMatrix(N_d, N_h);
	copyToDeviceMatrix(P_d, P_h);

    /* Copy mask to device constant memory */
    //INSERT CODE HERE
    hipMemcpyToSymbol(HIP_SYMBOL(M), M_h.elements, FILTER_SIZE * FILTER_SIZE * sizeof(float));

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    /*Launch kernel ---------------------------------------------------------*/
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);
    //INSERT CODE HERE
	unsigned TILE_SIZE = 16;
	unsigned BLOCK_SIZE = TILE_SIZE + FILTER_SIZE - 1;
	dim_block = dim3(BLOCK_SIZE, BLOCK_SIZE);
	dim_grid = dim3(ceil(N_h.width/(float)TILE_SIZE), ceil(N_h.height/(float)TILE_SIZE), 1);
	convolution <<< dim_grid, dim_block >>> (N_d, P_d);
    
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy data from device to host ----------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    copyFromDeviceMatrix(P_h, P_d);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(M_h, N_h, P_h);

    // Free memory ------------------------------------------------------------

     freeMatrix(M_h);
     freeMatrix(N_h);
     freeMatrix(P_h);
     freeDeviceMatrix(N_d);
     freeDeviceMatrix(P_d);

     return 0;
}

