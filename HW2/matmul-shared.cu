#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <sys/time.h>

#ifdef DEBUG
#define CUDA_CHECK(x) do {\
	(x);\
	hipError_t e = hipGetLastError();\
	if(hipSuccess !=e){\
		printf("cuda failure \"%s\" at %s:%d\n",\
			hipGetErrorString(e),\
			__FILE__,__LINE__);\
		exit(1);\
	}\
}while(0)
#else
#define CUDA_CHECK(x)	(x)
#endif

//CUDA kernel size settings
const int TILE_WIDTH = 32; // block will be (TILE_WIDTH,TILE_WIDTH)

//random data generation
void genData(float* ptr, unsigned int size) {
	while (size) {
		*ptr++ = (float)size/(float)1000;
		size--;
	}
}

__global__ void matmul(float* g_C, const float* g_A, const float* g_B, const int width1, const int width2, const int width3) {
	__shared__ float s_A[TILE_WIDTH][TILE_WIDTH];
	__shared__ float s_B[TILE_WIDTH][TILE_WIDTH];

	int by = blockIdx.y; int bx = blockIdx.x;
	int ty = threadIdx.y; int tx = threadIdx.x;

	int gy = by * TILE_WIDTH + ty; // global y index
	int gx = bx * TILE_WIDTH + tx; // global x index

	float sum = 0.0F;
	for (register int m = 0; m < ceil(width2 / (float)TILE_WIDTH); ++m) {
	
		// read into the shared memory blocks
		if((m * TILE_WIDTH + tx) >= width2 || gy >= width1)		s_A[ty][tx] = 0.0F;
		else													s_A[ty][tx] = g_A[gy * width2 + (m * TILE_WIDTH + tx)];
		if((m * TILE_WIDTH + ty) >= width2 || gx >= width3)		s_B[ty][tx] = 0.0F;
		else													s_B[ty][tx] = g_B[(m * TILE_WIDTH + ty) * width3 + gx];
		__syncthreads();

		for (register int k = 0; k < TILE_WIDTH; ++k) {
			sum += s_A[ty][k] * s_B[k][tx];
		}
		__syncthreads();
	}
	if(gy < width1 && gx < width3)	g_C[gy * width3 + gx] = sum;
}

int main(int argc, char* argv[]) {

	// host-side data
	const int WIDTH1 = atoi(argv[1]);
	const int WIDTH2 = atoi(argv[2]);
	const int WIDTH3 = atoi(argv[3]);
	const int GRID_WIDTH1 = ceil(WIDTH1 / (float)TILE_WIDTH); // grid will be (GRID_WDITH,GRID_WDITH)
	const int GRID_WIDTH2 = ceil(WIDTH2 / (float)TILE_WIDTH);
	const int GRID_WIDTH3 = ceil(WIDTH3 / (float)TILE_WIDTH); 

	float* pA = NULL;
	float* pB = NULL;
	float* pC = NULL;
	struct timeval start_time, end_time;

	// malloc memories on the host-side
	pA = (float*)malloc(WIDTH2 * WIDTH1 * sizeof(float));
	pB = (float*)malloc(WIDTH3 * WIDTH2 * sizeof(float));
	pC = (float*)malloc(WIDTH3 * WIDTH1 * sizeof(float));
	for(int i=0;i < WIDTH1 * WIDTH3;i++)	pC[i] = 0.0;

	// generate source data
	genData(pA, WIDTH2 * WIDTH1);
	genData(pB, WIDTH3 * WIDTH2);

	// CUDA: allocate device memory
	float* pAdev = NULL;
	float* pBdev = NULL;
	float* pCdev = NULL;
	CUDA_CHECK( hipMalloc((void**)&pAdev, WIDTH2 * WIDTH1 * sizeof(float)) );
	CUDA_CHECK( hipMalloc((void**)&pBdev, WIDTH3 * WIDTH2 * sizeof(float)) );
	CUDA_CHECK( hipMalloc((void**)&pCdev, WIDTH3 * WIDTH1 * sizeof(float)) );

	// copy from host to device
	CUDA_CHECK( hipMemcpy(pAdev, pA, WIDTH2 * WIDTH1 * sizeof(float), hipMemcpyHostToDevice) );
	CUDA_CHECK( hipMemcpy(pBdev, pB, WIDTH3 * WIDTH2 * sizeof(float), hipMemcpyHostToDevice) );

	//get current time
	hipDeviceSynchronize();
	gettimeofday(&start_time, NULL);

	// CUDA: launch the kernel
	dim3 dimGrid(GRID_WIDTH3, GRID_WIDTH1, 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	matmul <<< dimGrid, dimBlock >>> (pCdev, pAdev, pBdev, WIDTH1, WIDTH2, WIDTH3);
	CUDA_CHECK( hipPeekAtLastError() );

	//get current time
	hipDeviceSynchronize();
	gettimeofday(&end_time, NULL);
	double operating_time = (double)(end_time.tv_sec)+(double)(end_time.tv_usec)/1000000.0-((double)(start_time.tv_sec)+(double)(start_time.tv_usec)/1000000.0);
	printf("Elapsed: %f seconds\n", (double)operating_time);

	// copy from device to host
	CUDA_CHECK( hipMemcpy(pC, pCdev, WIDTH1 * WIDTH3 * sizeof(float), hipMemcpyDeviceToHost) );

	// free device memory
	CUDA_CHECK( hipFree(pAdev) );
	CUDA_CHECK( hipFree(pBdev) );
	CUDA_CHECK( hipFree(pCdev) );

	// print sample cases
	int i, j;
	for(i=0;i<WIDTH1;i++)
	{
		for(j=0;j<WIDTH3;j++)
			std::cout << pC[i*WIDTH3+j] << " ";
		std::cout << std::endl;
	}


	// done
	return 0;
}
